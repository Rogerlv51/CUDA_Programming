#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "freshman.h"

// 这是一个简单的两个向量相加的cuda程序

// cpu加法
void sumArrays(float * a,float * b,float * res,const int size)
{
  for(int i=0;i<size;i+=4)
  {
    res[i]=a[i]+b[i];
    res[i+1]=a[i+1]+b[i+1];
    res[i+2]=a[i+2]+b[i+2];
    res[i+3]=a[i+3]+b[i+3];
  }
}

// GPU加法
__global__ void sumArraysGPU(float*a,float*b,float*res)
{
  int i=threadIdx.x;
  res[i]=a[i]+b[i];
}
int main(int argc,char **argv)
{
  int dev = 0;
  hipSetDevice(dev);

  int nElem=32;
  printf("Vector size:%d\n",nElem);
  int nByte=sizeof(float)*nElem;
  float *a_h=(float*)malloc(nByte);
  float *b_h=(float*)malloc(nByte);
  float *res_h=(float*)malloc(nByte);
  float *res_from_gpu_h=(float*)malloc(nByte);
  memset(res_h,0,nByte);
  memset(res_from_gpu_h,0,nByte);

  float *a_d,*b_d,*res_d;
  CHECK(hipMalloc((float**)&a_d,nByte));
  CHECK(hipMalloc((float**)&b_d,nByte));
  CHECK(hipMalloc((float**)&res_d,nByte));
  /*
    分配设备端的内存空间，为了区分设备和主机端内存，我们可以给变量加后缀或者前缀h_表示host，d_表示device
    一个经常会发生的错误就是混用设备和主机的内存地址！！!
  */

  initialData(a_h,nElem);
  initialData(b_h,nElem);

  CHECK(hipMemcpy(a_d,a_h,nByte,hipMemcpyHostToDevice));
  CHECK(hipMemcpy(b_d,b_h,nByte,hipMemcpyHostToDevice));

  /*
    hipError_t hipMemcpy(void * dst,const void * src,size_t count, hipMemcpyKind kind)
    这个函数是内存拷贝过程，可以完成以下几种过程（hipMemcpyKind kind），从名称上可以很直观地看出内存拷贝的方向
        hipMemcpyHostToHost
        hipMemcpyHostToDevice
        hipMemcpyDeviceToHost
        hipMemcpyDeviceToDevice
    如果函数执行成功，则会返回 hipSuccess 否则返回 hipErrorOutOfMemory
    可以使用如下函数把错误信息翻译成详细信息：
        char* hipGetErrorString(hipError_t error)
  */

  dim3 block(nElem);
  dim3 grid(nElem/block.x);
  sumArraysGPU<<<grid,block>>>(a_d,b_d,res_d);
  printf("Execution configuration<<<%d,%d>>>\n",block.x,grid.x);

  CHECK(hipMemcpy(res_from_gpu_h,res_d,nByte,hipMemcpyDeviceToHost));
  sumArrays(a_h,b_h,res_h,nElem);

  checkResult(res_h,res_from_gpu_h,nElem);
  hipFree(a_d);
  hipFree(b_d);
  hipFree(res_d);

  free(a_h);
  free(b_h);
  free(res_h);
  free(res_from_gpu_h);

  return 0;
}